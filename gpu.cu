
#include <hip/hip_runtime.h>
#define N 1000000


__global__
void add(float *a, float *b) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	a[i] = 31.0f;
	b[i] = 0.43f;
	a[i] = a[i] + b[i];
}


int main() {
	float *a, *b;
	hipMallocManaged(&a, N * sizeof(float));
	hipMallocManaged(&b, N * sizeof(float));

	for(int j = 0; j < 10; j ++)
	add<<<N/256, 256>>>(a, b);
	hipDeviceSynchronize();

	hipFree(a);
	hipFree(b);
	return 0;
}

